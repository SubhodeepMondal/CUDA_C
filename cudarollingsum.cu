#include<iostream>
#include"cudalibrary.h"

int main(){

    freopen("cudarollingsum.csv","w",stdout);
    double *a,b;
    int i,n=16;

    hipSetDevice(0);

    hipMallocManaged((double**)&a, n * sizeof(double));
    hipMallocManaged((double**)&b, n * sizeof(double));

    for (i=0;i<n;i++)
        a[i] =(double) 13*i/32;

    b = 0;
    for (i=0;i<n;i++)
        b += a[i];
    

    //dim3 grid,block;
    dim3 block(n);
    dim3 grid(1,1);
    
    cudaRollingSum<<<grid,block>>>(a);
    hipDeviceSynchronize();

    std::cout << b << ", ";
    
    std::cout << a[0] << ",";
}
#include<iostream>
#include"cudalibrary.h"

int main(){

    freopen("io/cudarollingsum.csv","w",stdout);
    double *a,b;
    int i,n=16;

    hipSetDevice(0);

    hipMallocManaged((double**)&a, n * sizeof(double));
    hipMallocManaged((double**)&b, n * sizeof(double));

    for (i=0;i<n;i++)
        a[i] =(double) 13*i/32;

    b = 0;
    for (i=0;i<n;i++)
        b += a[i];

    for (i=0;i<n;i++)
        std::cout << a[i] << ",";

    std::cout << std::endl;
    

    //dim3 grid,block;
    dim3 block(n);
    dim3 grid(1,1);
    
    cudaRollingSum<<<grid,block>>>(a);
    hipDeviceSynchronize();



    std::cout << b << ", "<< std::endl;
    
    for (i=0;i<n;i++)
        std::cout << a[i] << ",";
}